
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define ROUNDS 1000000000
#define ROLLS 231
#define BLOCKSIZE 1024

__global__ void roll_cuda(int *results, unsigned long long seed, int numBlocks)
{
    
    int highest = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    int current;

    const size_t runs = (ROUNDS / (BLOCKSIZE * numBlocks))
                      + ((threadIdx.x + blockIdx.x * numBlocks) < (ROUNDS % (BLOCKSIZE * numBlocks)));
    for (size_t i = 0; i < runs; i++)
    {
        current = __popc(hiprand(&state) & hiprand(&state));
        current += __popc(hiprand(&state) & hiprand(&state));
        current += __popc(hiprand(&state) & hiprand(&state));
        current += __popc(hiprand(&state) & hiprand(&state));
        current += __popc(hiprand(&state) & hiprand(&state));
        current += __popc(hiprand(&state) & hiprand(&state));
        current += __popc(hiprand(&state) & hiprand(&state));
        current += __popc(hiprand(&state) & hiprand(&state) << 25);
        highest = max(highest, current);
    }
    results[idx] = highest;
}

int main()
{
    hipDeviceProp_t prop;
    int deviceId;
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&prop, deviceId);
    int smCount = prop.multiProcessorCount;
    
    int highest = 0;
    int *results;
    
    
    int maxActiveBlocks = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, roll_cuda, BLOCKSIZE, 0);
    int numBlocks = smCount * maxActiveBlocks;
    hipMallocManaged(&results, numBlocks * BLOCKSIZE * sizeof(*results));
    
    float totalTime=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    roll_cuda<<<numBlocks, BLOCKSIZE>>>(results, time(NULL), numBlocks);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&totalTime, start, stop);

    for (size_t i = 0; i < numBlocks * BLOCKSIZE; i++)
    {
        highest = std::max(highest, results[i]);
    }

    std::cout << "My record is: " << (int)highest << ".\nIt took me " << totalTime << "ms.\n";
    return 0;
}
